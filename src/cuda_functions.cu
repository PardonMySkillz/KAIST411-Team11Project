
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define CEIL_DIV(X, Y) (((X)+(Y)-1)/(Y))

extern "C"{

float* np2cuda(float* input, int size){
    float* output;

    hipMalloc((void**)&output, sizeof(float) * size);
	hipMemcpy(output, input, sizeof(float) * size, hipMemcpyHostToDevice);

    return output;
}

float* cuda2np(float* input, int size){
    float* output = (float*) malloc(sizeof(float) * size);
    hipMemcpy(output, input, sizeof(float) * size, hipMemcpyDeviceToHost);
    
    return output;
}

void cuda_free(float* input){
    hipFree(input);
}

void c_free(float* input){
    free(input);
}

void block_cpu(){
    hipEvent_t block;
    hipEventCreateWithFlags(&block, hipEventBlockingSync);
    hipEventRecord(block);
    hipEventSynchronize(block);
    hipEventDestroy(block);
}


// TODOs
// implement functions whose functionality complies with restricted PyTorch functions
// There are two type of functions to implement for a single operation:
//   CUDA function
//   Interface function that calls CUDA function
// Note that interface function gets the float pointer already malloced at GPU
__global__ void _leaky_relu(float* input, float* output, int size, float negative_slope){

    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < size) output[index] = input[index] < 0 ? input[index] * negative_slope : input[index];

}
float* leaky_relu(float* input, int height, int width, float negative_slope){

    float *device_input, *device_output;
    unsigned long size = height * width;

    hipMalloc((void **) &device_input, size * sizeof(float));
    hipMalloc((void **) &device_output, size * sizeof(float));

    hipMemcpy(device_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(device_output, 0, size * sizeof(float));

    int threadsPerBlock = 256;

    _leaky_relu<<<CEIL_DIV(size, threadsPerBlock), threadsPerBlock>>>(device_input, device_output, size, negative_slope);

    return device_output;

}

__global__ void _batch_norm(float* input, float* output, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){

    uint b = blockIdx.x;
    uint c = blockIdx.y;

    uint idx = blockIdx.z * 128 + threadIdx.x;

    uint io_index = b * channels * height * width + c * height * width + idx;

    float e = 1e-5;
    if(idx < width * height)
        output[io_index] = weight[c] * ((input[io_index] - running_mean[c]) / sqrt(running_var[c] + e)) + bias[c];

}
float* batch_norm(float* input, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){
    float *device_output;
    unsigned long io_size = batch_size * channels * height * width * sizeof(float);

    hipMalloc((void**) &device_output, io_size);

    int threadsPerBlock = 128;
    dim3 numBlocks(batch_size, channels, CEIL_DIV(height * width, threadsPerBlock));

    _batch_norm<<<numBlocks, threadsPerBlock>>>(input, device_output, channels, height, width, running_mean, running_var, weight, bias);

    hipDeviceSynchronize();

    return device_output;

}

__global__ void _conv2d(int batch_size, float* input, int input_channels, int input_height, int input_width,
                              float* weight, float* bias, int kernel_height, int kernel_width,
                              int output_channel, int output_height, int output_width, int stride, float* output) {
    int batch = blockIdx.x;
    // int out_c = blockIdx.y;
    // int out_h = threadIdx.x;
    // int out_w = threadIdx.y;
    int out_c = threadIdx.x;
    // for (int batch = 0; batch < batch_size; batch++){
    // for (int out_c = 0; out_c < output_channel; out_c++){
    for (int out_h = 0; out_h < output_height; out_h++){
        for (int out_w = 0; out_w < output_width; out_w++){
            int i_h_start = out_h * stride;
            int i_w_start = out_w * stride;
            float result = 0.0;
            if (bias != NULL) {
                result += bias[out_c];
            }
            for (int kernel_h = 0; kernel_h < kernel_height; kernel_h++) {
                for (int kernel_w = 0; kernel_w < kernel_width; kernel_w++) {
                    for (int in_c = 0; in_c < input_channels; in_c++) {
                        int i_h = i_h_start + kernel_h;
                        int i_w = i_w_start + kernel_w;
                        if (i_h >= 0 && i_h < input_height && i_w >= 0 && i_w < input_width) {
                            int input_index = batch * input_channels * input_height * input_width + in_c * input_height * input_width + i_h * input_width + i_w;
                            int kernel_index = out_c * input_channels * kernel_height * kernel_width + in_c * kernel_height * kernel_width + kernel_h * kernel_width + kernel_w;
                            result += input[input_index] * weight[kernel_index];

                        }
                    }
                }
            }
            int output_index = batch * output_channel * output_height * output_width + out_c * output_height * output_width + out_h * output_width + out_w;
            output[output_index] = result;
            // if (output_index < 10){
            //     printf("index: %d; result: %f", output_index, result);
            // }    
            
        }
    }
    // }
    // }
}
float* conv2d(int batch_size, float* input, int input_channels, int input_height, int input_width,
              float* weight, float* bias, int kernel_height, int kernel_width,
              int output_channel, int output_height, int output_width, int stride){
    int output_size = batch_size * output_channel * output_height * output_width;
    float* d_input;
    float* d_weight;
    float* d_bias;
    float* d_output;
    // for (int i = 0; i < 10; i++){
    //     printf("aaaaaaaaa %f\n", input[i]);
    // }
    // Allocate device memory
    hipMalloc((void**)&d_input, input_channels * input_height * input_width * batch_size * sizeof(float));
    hipMalloc((void**)&d_weight, input_channels * kernel_height * kernel_width * output_channel * sizeof(float));
    hipMalloc((void**)&d_bias, output_channel * sizeof(float));
    hipMalloc((void**)&d_output, output_size * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, input, input_channels * input_height * input_width * batch_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, input_channels * kernel_height * kernel_width * output_channel * sizeof(float), hipMemcpyHostToDevice);
    if (bias != NULL){
    hipMemcpy(d_bias, bias, output_channel * sizeof(float), hipMemcpyHostToDevice);
    }
    else{
        d_bias = NULL;
    }
    // Configure grid and block dimensions
    dim3 grid(batch_size, output_channel);
    dim3 block(output_height, output_width);
    
    // Launch the CUDA kernel
    _conv2d<<<batch_size, output_channel>>>(batch_size, d_input, input_channels, input_height, input_width, 
                    d_weight, d_bias, kernel_height, kernel_width,
                    output_channel, output_height, output_width, stride, d_output);
    
    // Allocate memory for the output on the host
    float* output = (float*)malloc(output_size * sizeof(float));
    if (output == NULL) {
        // Handle memory allocation error
        return NULL;
    }

    // Copy the result from device to host
    // cudaMemcpy(output, d_output, output_size * sizeof(float), cudaMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    // cudaFree(d_output);
    // for (int i = 0; i < 10; i++){
    //     printf("aaaaaaaaa %f\n", output[i]);
    // }
    return d_output;


}


__global__ void _max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride, float* output, int output_height, int output_width) {
    int batch = blockIdx.x;
    int channel = threadIdx.x;
    for (int row = 0; row < output_height; row++) {
        for (int col = 0; col < output_width; col++) {
            int start_row = row * stride;
            int start_col = col * stride;
            float max_value = input[batch * input_channel * input_height * input_width + channel * input_height *input_width + start_row * input_width + start_col];
            for (int i =0; i < kernel_height; i++) {
                for (int j = 0; j < kernel_width; j++) {
                    float curr_value = input[batch * input_channel * input_height * input_width + channel * input_height *input_width + (start_row + i) * input_width + start_col + j];
                    if (curr_value > max_value) {
                        max_value = curr_value;
                    }
                }
            }
            output[batch * input_channel * output_height * output_width + channel * output_height * output_width + row * output_width + col] = max_value;
        }
    }
}

float* max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride){
    float* d_input, *d_output;

    int output_height = floor((input_height - kernel_height) / stride) + 1;
    int output_width = floor((input_width - kernel_width) / stride) + 1;
    int output_size = batch_size * input_channel * output_height * output_width;

    hipMalloc((void**)&d_input, input_channel * input_height * input_width * batch_size * sizeof(float));
    hipMalloc((void**)&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, input, input_channel * input_height * input_width * batch_size * sizeof(float), hipMemcpyHostToDevice);



    _max_pool2d<<<batch_size, input_channel>>>(batch_size, d_input, input_channel, input_height, input_width, kernel_height, kernel_width, stride, d_output, output_height, output_width);

    

    hipFree(d_input);
    return d_output;
}

// Unused
__global__ void _pad(float *input, float* output, int batch_size, int channels, int height, int width, int left, int right, int top, int bottom, float padding) {
    // int new_height = height + top + bottom;
    int new_width = width + left + right;

    float *ptri = input;
    float *ptro = output;
    for (int b = 0; b < batch_size; b++)
        for (int c = 0; c < channels; c++)
        {
            // Pad the top
            for (int i = 0; i < top * new_width; i++)
                ptro[i] = padding;

            // Pad the middle
            for (int i = 0; i < height; i++)
            {
                // Left
                for (int j = 0; j < left; j++, ptro++)
                    *ptro = padding;
                //
                for (int j = 0; j < height; j++, ptri++, ptro++)
                    *ptro = *ptri;
                // Right
                for (int j = 0; j < right; j++, ptro++)
                    *ptro = padding;
            }

            // Pad the end
            for (int i = 0; i < bottom * new_width; i++, ptro++)
                *ptro = padding;
        }
}

__global__ void _pad_fill(float* arr, int size, float value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = idx; i < size; i += gridDim.x * blockDim.x)
        arr[i] = value;
}

float *pad(float *input_ptr, int batch_size, int channels, int height, int width, int left, int right, int top, int bottom, float padding)
{
    float *d_output;
    int new_height = height + top + bottom;
    int new_width = width + left + right;

    // int input_size = batch_size * channels * height * width;
    int output_size =  batch_size * channels * new_height * new_width;

    hipMalloc((void **)&d_output, sizeof(float) * output_size);

    int blockSize = 256;
    int numBlocks = (output_size + blockSize - 1) / blockSize;
    _pad_fill<<<numBlocks, blockSize>>>(d_output, output_size, padding);
    hipDeviceSynchronize();
    
    
    for (int b = 0; b < batch_size; b++)
        for (int c = 0; c < channels; c++)
        {
            int old_offset
                = b * channels * height * width 
                + c * height * width;
            int new_offset 
                = b * channels * new_height * new_width 
                + c * new_height * new_width
                + top * new_width;
            for(int i = 0; i < height; i++)
                hipMemcpyAsync(d_output + new_offset + i * new_width + left, input_ptr + old_offset + i * width, width * sizeof(float), hipMemcpyDeviceToDevice);
        }

    hipDeviceSynchronize();

    return d_output;
}

}