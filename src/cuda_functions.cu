
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CEIL_DIV(X, Y) (((X)+(Y)-1)/(Y))

extern "C"{

float* np2cuda(float* input, int size){
    float* output;

    hipMalloc((void**)&output, sizeof(float) * size);
	hipMemcpy(output, input, sizeof(float) * size, hipMemcpyHostToDevice);

    return output;
}

float* cuda2np(float* input, int size){
    float* output = (float*) malloc(sizeof(float) * size);
    hipMemcpy(output, input, sizeof(float) * size, hipMemcpyDeviceToHost);
    
    return output;
}

void cuda_free(float* input){
    hipFree(input);
}

void c_free(float* input){
    free(input);
}

void block_cpu(){
    hipEvent_t block;
    hipEventCreateWithFlags(&block, hipEventBlockingSync);
    hipEventRecord(block);
    hipEventSynchronize(block);
    hipEventDestroy(block);
}


// TODOs
// implement functions whose functionality complies with restricted PyTorch functions
// There are two type of functions to implement for a single operation:
//   CUDA function
//   Interface function that calls CUDA function
// Note that interface function gets the float pointer already malloced at GPU
__global__ void _leaky_relu(float* input, float* output, int batch_size, int channels, int height, int width, int negative_slope){

    uint batch = blockIdx.x;
    uint channel = blockIdx.y;

    uint row = threadIdx.x;
    uint col = threadIdx.y;

    uint index = batch * channels * height * width + channel * height * width + row * width + col;
    
    if(input[index] < 0) {
        output[index] = negative_slope * input[index];
    } else {
        output[index] = input[index];
    }

}
float* leaky_relu(float* input, int batch_size, int channels, int height, int width, int negative_slope){

    float* output, *device_input, *device_output;
    unsigned long size = batch_size * channels * height * width;

    hipMalloc((void **) &device_input, size * sizeof(float));
    hipMalloc((void **) &device_output, size * sizeof(float));

    hipMemcpy(device_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(device_output, 0, size * sizeof(float));

    dim3 numBlocks(batch_size, channels);
    dim3 threadsPerBlock(height, width);

    _leaky_relu<<<numBlocks, threadsPerBlock>>>(device_input, device_output, batch_size, channels, height, width, negative_slope);

    hipMemcpy(output, device_output, size * sizeof(float), hipMemcpyDeviceToHost);
    cuda_free(device_input);
    cuda_free(device_output);

    return output;

}

__global__ void _batch_norm(float* input, float* output, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){

    uint batch = blockIdx.x;
    uint channel = blockIdx.y;

    uint row = threadIdx.x;
    uint col = threadIdx.y;

    uint io_index = batch * channels * height * width + channel * height * width + row * width + col;

    float e = 1e-5;

    output[io_index] = weight[channel] * ((input[io_index] - running_mean[channel]) / (running_var[channel] + e)) + bias[channel];

}
float* batch_norm(float* input, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){
    float* output, *device_input, *device_output;
    float *d_running_mean, *d_running_var, *d_weight, *d_bias;
    unsigned long io_size = batch_size * channels * height * width * sizeof(float);
    unsigned long mv_size = batch_size * channels * width * sizeof(float);

    hipMalloc((void**) &device_input, io_size);
    hipMalloc((void**) &device_output, io_size);
    hipMemcpy(device_input, input, io_size, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_running_mean, mv_size);
    hipMalloc((void**) &d_running_var, mv_size);
    hipMalloc((void**) &d_weight, mv_size);
    hipMalloc((void**) &d_bias, mv_size);
    hipMemcpy(d_running_mean, running_mean, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_running_var, running_var, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, mv_size, hipMemcpyHostToDevice);

    dim3 numBlocks(batch_size, channels);
    dim3 threadsPerBlock(height, width);

    _batch_norm<<<numBlocks, threadsPerBlock>>>(device_input, device_output, batch_size, channels, height, width, d_running_mean, d_running_var, d_weight, d_bias);

    hipMemcpy(output, device_output, io_size, hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);

    hipFree(d_running_mean);
    hipFree(d_running_var);
    hipFree(d_bias);
    hipFree(d_weight);
    return output;

}

__global__ void _conv2d(){}
float* conv2d(){}

__global__ void _max_pool2d(float* input, int input_height, int input_width, int kernel_width, int kernel_height, int stride, float* output, int output_width, int output_height){
    const uint col = blockIdx.x * blockDim.x + threadIdx.x;
    const uint row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col < output_width && row < output_height) {
        int start_row = row * stride;
        int start_col = col * stride;
        float max_value = input[start_row * input_width + start_col];
        for (int i = 0; i < kernel_height; i++) {
            for (int j = 0; j < kernel_width; j++) {
                float curr_value = input[(start_row+i)*input_width+(start_col+j)];
                if (curr_value > max_value) {
                    max_value = curr_value;
                }
            }
        }
        output[row*output_width+col] = max_value;
    }
    
}
float* max_pool2d(float* input, int input_height, int input_width, int kernel_width, int kernel_height, int stride){
    float* output, *device_input, *device_output;
    int output_height = floor((input_height-kernel_height)/stride) +1;
    int output_width = floor((input_width-kernel_width)/stride) +1;
    int output_size = output_height*output_width;
    int input_size = input_height* input_width*sizeof(float);

    hipMalloc((void**)&device_input, input_size);
    hipMalloc((void**)&device_output, output_size * sizeof(float));

    hipMemcpy(device_input, input, input_size, hipMemcpyHostToDevice);
    hipMemset(device_output, 0, output_size*sizeof(float));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_width + threadsPerBlock.x - 1) / threadsPerBlock.x, (output_height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    _max_pool2d<<<numBlocks, threadsPerBlock>>>(device_input, input_height, input_width, kernel_height, kernel_width, stride, device_output, output_width, output_height);
    hipMemcpy(output, device_output,output_size, hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);
    return output;
}

__global__ void _pad(){}
float* pad(){}

}