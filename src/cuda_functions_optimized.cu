
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

#define CEIL_DIV(X, Y) (((X)+(Y)-1)/(Y))

#include <time.h>
unsigned long long get_time_ns()
{
    struct timespec ts;

    if (clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &ts) == -1)
    {
        printf("clock_gettime error\n");
        exit(-1);
    }

    return ts.tv_sec * 1000000000ULL + ts.tv_nsec;
}

extern "C"{

// TODOs
// implement CUDA optimized functions whose functionality complies with restricted PyTorch functions
__global__ void _leaky_relu(float* input, float* output, int size, float negative_slope){

    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < size)
        output[index] = input[index] * (input[index] < 0 ? negative_slope: 1);
}
float* leaky_relu(float* input, int height, int width, float negative_slope){

    float *device_output;
    unsigned long size = height * width;

    hipMalloc((void **) &device_output, size * sizeof(float));

    int threadsPerBlock = 1024;

    _leaky_relu<<<CEIL_DIV(size, threadsPerBlock), threadsPerBlock>>>(input, device_output, size, negative_slope);

    return device_output;

}

__global__ void _batch_norm(float *input, float *output, int channels, int sz2d, float *running_mean, float *running_var, float *weight, float *bias)
{

    uint b = blockIdx.x;
    uint c = blockIdx.y;

    uint idx = blockIdx.z * blockDim.x + threadIdx.x;

    uint io_index = b * channels * sz2d + c * sz2d + idx;

    float e = 1e-5;
    if (idx < sz2d)
        output[io_index] = weight[c] * ((input[io_index] - running_mean[c]) / __fsqrt_rd(running_var[c] + e)) + bias[c];
}

float* batch_norm(float* input, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){
    float *device_output;
    int sz2d = height * width;
    unsigned long io_size = batch_size * channels * sz2d * sizeof(float);

    hipMalloc((void**) &device_output, io_size);

    int threadsPerBlock = 128;
    dim3 numBlocks(batch_size, channels, CEIL_DIV(sz2d, threadsPerBlock));

    _batch_norm<<<numBlocks, threadsPerBlock>>>(input, device_output, channels, sz2d, running_mean, running_var, weight, bias);

    hipDeviceSynchronize();

    return device_output;

}

__global__ void _conv2d(){}
float* conv2d(){}

__global__ void _max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride, float* output, int output_height, int output_width){
    int batch = blockIdx.x;
    int channel = blockIdx.y;
    int row = threadIdx.x;
    int col = threadIdx.y;
    

    int start_row = row * stride;
    int start_col = col * stride;
    float max_value = input[batch * input_channel * input_height * input_width + channel * input_height * input_width + start_row * input_width + start_col];
    for (int i=0; i < kernel_height; i++) {
        for (int j=0; j < kernel_width; j++) {
            float curr_value = input[batch * input_channel * input_height * input_width + channel * input_height * input_width + (start_row + i) * input_width + start_col + j];
            if (curr_value > max_value) {
                max_value = curr_value;
            }
        }
    }
    output[batch * input_channel * output_height * output_width + channel * output_height * output_width + row * output_width + col] = max_value;
}
float* max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride){
    float* d_input, *d_output;

    int output_height = floor((input_height - kernel_height) / stride) + 1;
    int output_width = floor((input_width - kernel_width) / stride) + 1;
    int output_size = batch_size * input_channel * output_height * output_width;

    hipMalloc((void**)&d_input, input_channel * input_height * input_width * batch_size * sizeof(float));
    hipMalloc((void**)&d_output, output_size * sizeof(float));
    hipMemcpy(d_input, input, input_channel * input_height * input_width * batch_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadPerBlock(output_height, output_width);
    dim3 numBlocks(batch_size, input_channel);
    _max_pool2d<<<numBlocks, threadPerBlock>>>(batch_size, d_input, input_channel, input_height, input_width, kernel_height, kernel_width, stride, d_output, output_height, output_width);
    hipFree(d_input);
    return d_output;
}

__global__ void _pad(float *input, float *output, int size, int height, int width, int left, int right, int top, int bottom, int sz2d, float padding)
{
    int new_height = height + top + bottom;
    int new_width = width + left + right;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) return;
    int bc = idx / sz2d;
    int off = idx % sz2d;
    int y = (off / new_width) - top;
    int x = (off % new_width) - left;

    float *bci = input + bc * height * width;
    float *bco = output + bc * new_height * new_width + new_width * top + left;

    if (x < 0 || width <= x || y < 0 || height <= y)
        output[idx] = padding;
    else
        output[idx] = bci[width * y + x];
}

float *pad(float *input_ptr, int batch_size, int channels, int height, int width, int left, int right, int top, int bottom, float padding)
{
    float *d_input = input_ptr, *d_output;
    int new_height = height + top + bottom;
    int new_width = width + left + right;

    // int input_size = batch_size * channels * height * width;
    int output_size = batch_size * channels * new_height * new_width;

    int batchannels = batch_size * channels;

#ifdef DEBUG
    printf("Allocating %f MB...\n", (float)(output_size) / 1e6 * sizeof(float));
#endif

    hipMalloc((void **)&d_output, sizeof(float) * output_size);

    // Fill array with padding
    int block_size = 1024;
    int num_blocks = CEIL_DIV(output_size, block_size);
    
    _pad<<<num_blocks, block_size>>>(d_input, d_output, output_size, height, width, left, right, top, bottom, new_height * new_width, padding);

    hipDeviceSynchronize();

    return d_output;
}
}