
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CEIL_DIV(X, Y) (((X)+(Y)-1)/(Y))

extern "C"{

// TODOs
// implement CUDA optimized functions whose functionality complies with restricted PyTorch functions
__global__ void _leaky_relu(float* input, float* output, int height, int width, float negative_slope){


    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    
    output[index] = input[index] < 0 ?  negative_slope * input[index] : input[index];

}
float* leaky_relu(float* input, int height, int width, float negative_slope){

    float *device_input, *device_output;
    unsigned long size =  height * width;

    hipMalloc((void **) &device_input, size * sizeof(float));
    hipMalloc((void **) &device_output, size * sizeof(float));

    hipMemcpy(device_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(device_output, 0, size * sizeof(float));

    // dim3 threadsPerBlock(16);
    // dim3 numBlock((height * width +threadsPerBlock.x - 1)/ threadsPerBlock.x);

    // _leaky_relu<<<numBlock, threadsPerBlock>>>(device_input, device_output, height, width, negative_slope);

    dim3 threadsPerBlock(height * width);

    _leaky_relu<<<1, threadsPerBlock>>>(device_input, device_output, height, width, negative_slope);

    hipDeviceSynchronize();

    hipFree(device_input);

    return device_output;

}

__global__ void _batch_norm(float* input, float* output, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){

    uint batch = blockIdx.x;
    uint channel = blockIdx.y;

    uint row = threadIdx.x;
    uint col = threadIdx.y;

    uint io_index = batch * channels * height * width + channel * height * width + row * width + col;

    float e = 1e-5;
    output[io_index] = weight[channel] * ((input[io_index] - running_mean[channel]) / sqrt(running_var[channel] + e)) + bias[channel];

}
float* batch_norm(float* input, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){
    float* output, *device_input, *device_output;
    float *d_running_mean, *d_running_var, *d_weight, *d_bias;
    unsigned long io_size = batch_size * channels * height * width * sizeof(float);
    unsigned long mv_size = channels*sizeof(float);


    output = (float*)malloc(io_size);

    hipMalloc((void**) &device_input, io_size);
    hipMalloc((void**) &device_output, io_size);
    hipMemcpy(device_input, input, io_size, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_running_mean, mv_size);
    hipMalloc((void**) &d_running_var, mv_size);
    hipMalloc((void**) &d_weight, mv_size);
    hipMalloc((void**) &d_bias, mv_size);
    hipMemcpy(d_running_mean, running_mean, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_running_var, running_var, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, mv_size, hipMemcpyHostToDevice);

    dim3 numBlocks(batch_size, channels);
    dim3 threadsPerBlock(height, width);

    _batch_norm<<<numBlocks, threadsPerBlock>>>(device_input, device_output, batch_size, channels, height, width, d_running_mean, d_running_var, d_weight, d_bias);

    hipDeviceSynchronize();

    hipFree(device_input);
    hipFree(d_running_mean);
    hipFree(d_running_var);
    hipFree(d_bias);
    hipFree(d_weight);

    return device_output;

}

__global__ void _conv2d(){}
float* conv2d(){}

__global__ void _max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride, float* output){
    // Commented out for bugs - Aziz
    // const uint col = blockIdx.x * blockDim.x + threadIdx.x;
    // const uint row = blockIdx.y * blockDim.y + threadIdx.y;
    
    
    // if (col < input_width && row < input_height) {
    //     for (int b = 0; b < batch_size; b++) {
    //         for (int c=0; c < input_channel; c++) {                
    //             int start_row = row * stride;
    //             int start_col = col * stride;

    //             float max_value = input[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + (start_row * input_width) + start_col];
    //             __shared__ float shared_input[kernel_height * kernel_width]; //shared memory for input
    //             shared_input[threadIdx.y * kernel_width + threadIdx.x] = input[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + ((start_row + threadIdx.y) * input_width) + (start_col + threadIdx.x)];
    //             __syncthreads();

    //             for (int i = 0; i < kernel_height; i++) {
    //                 for (int j = 0; j < kernel_width; j++) {
    //                     float curr_value = input[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + ((start_row + i) * input_width) + (start_col + j)];
    //                     if (curr_value > max_value) {
    //                         max_value = curr_value;
    //                     }
    //                 }
    //             }
    //             output[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + (row * input_width) + col] = max_value;

    //         }
    //     }
    // }
}
float* max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride){
    float* output, *device_output, *device_input;
    int size = batch_size * input_channel * input_height * input_width;
    hipMalloc((void**)&device_input, size * sizeof(float));
    hipMalloc((void**)&device_output, size * sizeof(float));

    hipMemcpy(device_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(device_output, 0, size * sizeof(float));
    dim3 threadPerBlock(32, 32);
    dim3 numBlocks((input_width + threadPerBlock.x - 1) / threadPerBlock.x, (input_height + threadPerBlock.y - 1) / threadPerBlock.y);
    _max_pool2d<<<numBlocks, threadPerBlock>>>(batch_size, device_input, input_channel, input_height, input_width, kernel_height, kernel_width, stride, device_output);
    
    output = (float*)malloc(size * sizeof(float));
    hipMemcpy(output, device_output, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);
    return output;
}

__global__ void _pad(){}
float* pad(){}

}