#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CEIL_DIV(X, Y) (((X)+(Y)-1)/(Y))

extern "C"{

// TODOs
// implement CUDA optimized functions whose functionality complies with restricted PyTorch functions
__global__ void _leaky_relu(){}
float* leaky_relu(){}

__global__ void _batch_norm(){}
float* batch_norm(){}

__global__ void _conv2d(){}
float* conv2d(){}

__global__ void _max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride, float* output){
    const uint col = blockIdx.x * blockDim.x + threadIdx.x;
    const uint row = blockIdx.y * blockDim.y + threadIdx.y;
    
    
    if (col < input_width && row < input_height) {
        for (int b = 0; b < batch_size; b++) {
            for (int c=0; c < input_channel; c++) {                
                int start_row = row * stride;
                int start_col = col * stride;

                float max_value = input[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + (start_row * input_width) + start_col];
                __shared__ float shared_input[kernel_height * kernel_width]; //shared memory for input
                shared_input[threadIdx.y * kernel_width + threadIdx.x] = input[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + ((start_row + threadIdx.y) * input_width) + (start_col + threadIdx.x)];
                __syncthreads();

                for (int i = 0; i < kernel_height; i++) {
                    for (int j = 0; j < kernel_width; j++) {
                        float curr_value = input[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + ((start_row + i) * input_width) + (start_col + j)];
                        if (curr_value > max_value) {
                            max_value = curr_value;
                        }
                    }
                }
                output[(b * input_channel * input_height * input_width) + (c * input_height * input_width) + (row * input_width) + col] = max_value;

            }
        }
    }
}
float* max_pool2d(int batch_size, float* input, int input_channel, int input_height, int input_width, int kernel_height, int kernel_width, int stride){
    float* output, *device_output, *device_input;
    int size = batch_size * input_channel * input_height * input_width;
    hipMalloc((void**)&device_input, size * sizeof(float));
    hipMalloc((void**)&device_output, size * sizeof(float));

    hipMemcpy(device_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(device_output, 0, size * sizeof(float));
    dim3 threadPerBlock(32, 32);
    dim3 numBlocks((input_width + threadPerBlock.x - 1) / threadPerBlock.x, (input_height + threadPerBlock.y - 1) / threadPerBlock.y);
    _max_pool2d<<<numBlocks, threadPerBlock>>>(batch_size, device_input, input_channel, input_height, input_width, kernel_height, kernel_width, stride, device_output);
    
    output = (float*)malloc(size * sizeof(float));
    hipMemcpy(output, device_output, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);
    return output;
}

__global__ void _pad(){}
float* pad(){}

}