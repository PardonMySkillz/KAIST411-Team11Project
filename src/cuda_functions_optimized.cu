
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CEIL_DIV(X, Y) (((X)+(Y)-1)/(Y))

extern "C"{

// TODOs
// implement CUDA optimized functions whose functionality complies with restricted PyTorch functions
__global__ void _leaky_relu(float* input, float* output, int batch_size, int channels, int height, int width, int negative_slope){

    uint batch = blockIdx.x;
    uint channel = blockIdx.y;

    uint row = threadIdx.x;
    uint col = threadIdx.y;

    uint index = batch * channels * height * width + channel * height * width + row * width + col;
    
    if(input[index] < 0) {
        output[index] = negative_slope * input[index];
    } else {
        output[index] = input[index];
    }

}
float* leaky_relu(float* input, int batch_size, int channels, int height, int width, int negative_slope){

    float* output, *device_input, *device_output;
    unsigned long size = batch_size * channels * height * width;

    hipMalloc((void **) &device_input, size * sizeof(float));
    hipMalloc((void **) &device_output, size * sizeof(float));

    hipMemcpy(device_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(device_output, 0, size * sizeof(float));

    dim3 numBlocks(batch_size, channels);
    dim3 threadsPerBlock(height, width);

    _leaky_relu<<<numBlocks, threadsPerBlock>>>(device_input, device_output, batch_size, channels, height, width, negative_slope);

    hipMemcpy(output, device_output, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);

    return output;

}

__global__ void _batch_norm(float* input, float* output, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){

    uint batch = blockIdx.x;
    uint channel = blockIdx.y;

    uint row = threadIdx.x;
    uint col = threadIdx.y;

    uint io_index = batch * channels * height * width + channel * height * width + row * width + col;

    float e = 1e-5;

    output[io_index] = weight[channel] * ((input[io_index] - running_mean[channel]) / (running_var[channel] + e)) + bias[channel];

}
float* batch_norm(float* input, int batch_size, int channels, int height, int width, float* running_mean, float* running_var, float* weight, float* bias){
    float* output, *device_input, *device_output;
    float *d_running_mean, *d_running_var, *d_weight, *d_bias;
    unsigned long io_size = batch_size * channels * height * width * sizeof(float);
    unsigned long mv_size = batch_size * channels * width * sizeof(float);

    hipMalloc((void**) &device_input, io_size);
    hipMalloc((void**) &device_output, io_size);
    hipMemcpy(device_input, input, io_size, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_running_mean, mv_size);
    hipMalloc((void**) &d_running_var, mv_size);
    hipMalloc((void**) &d_weight, mv_size);
    hipMalloc((void**) &d_bias, mv_size);
    hipMemcpy(d_running_mean, running_mean, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_running_var, running_var, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, mv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, mv_size, hipMemcpyHostToDevice);

    dim3 numBlocks(batch_size, channels);
    dim3 threadsPerBlock(height, width);

    _batch_norm<<<numBlocks, threadsPerBlock>>>(device_input, device_output, batch_size, channels, height, width, d_running_mean, d_running_var, d_weight, d_bias);

    hipMemcpy(output, device_output, io_size, hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);

    hipFree(d_running_mean);
    hipFree(d_running_var);
    hipFree(d_bias);
    hipFree(d_weight);
    return output;

}

__global__ void _conv2d(){}
float* conv2d(){}

__global__ void _max_pool2d(){}
float* max_pool2d(){}

__global__ void _pad(){}
float* pad(){}

}