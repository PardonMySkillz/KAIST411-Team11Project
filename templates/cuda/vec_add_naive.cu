
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 4096*4096

__global__ void cuda_vec_mul(float* out, float* a, float* b, int n){
	for(int i=0; i<n; ++i){
		out[i] = a[i]*b[i];
	}
}

int main() {
	float *a, *b, *out;
	size_t bytes = sizeof(float) * N;

	a = (float*) malloc(bytes);
	b = (float*) malloc(bytes);
	out = (float*) malloc(bytes);

	for(int i = 0; i < N; ++i){
		a[i] = 1.0;
		b[i] = 2.0;
	}
	
	// cudaMalloc(void **devPtr, size_t count);
	// cudaFree(void *devPtr);
	// cudaMemcpy(void *dst, void *src, size_t count, cudaMemcpyKind kind);
	// kind canbe cudaMemcpyHostToDevice or cudaMemcpyDeviceToHost
	
	float *da, *db, *dout;
	
	hipMalloc((void**)&da, bytes);
	hipMalloc((void**)&db, bytes);
	hipMalloc((void**)&dout, bytes);

	hipMemcpy(da, a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(db, b, bytes, hipMemcpyHostToDevice);
	cuda_vec_mul<<<1,1>>>(dout, da, db, N);

	hipMemcpy(out, dout, bytes, hipMemcpyDeviceToHost);
	
	printf("%f\n", out[3]); // expect 2.

	hipFree(da);
	hipFree(db);
	hipFree(dout);

	free(a);
	free(b);
	free(out);
}
