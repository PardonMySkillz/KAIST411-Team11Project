
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    printf("Hello World from Host!\n");
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
